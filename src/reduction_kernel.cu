#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <iostream>

#include "reduction.h"

#define SM_SIZE 1024

using namespace std;

void allocateDeviceMemory(void** M, int size)
{
    hipError_t err = hipMalloc(M, size);
    assert(err==hipSuccess);
}


void deallocateDeviceMemory(void* M)
{
    hipError_t err = hipFree(M);
    assert(err==hipSuccess);
}

void cudaMemcpyToDevice(void* dst, void* src, int size) {
    hipError_t err = hipMemcpy((void*)dst, (void*)src, size, hipMemcpyHostToDevice);
    assert(err==hipSuccess);
}

void cudaMemcpyToHost(void* dst, void* src, int size) {
    hipError_t err = hipMemcpy((void*)dst, (void*)src, size, hipMemcpyDeviceToHost);
    assert(err==hipSuccess);
}

void reduce_ref(const int* const g_idata, int* const g_odata, const int n) {
    for (int i = 0; i < 2048; i++)
        g_odata[0] += g_idata[i];
}

__global__ 
void reduce1(const int* const g_idata, int* const g_odata)
{
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    for (unsigned int s=1; s < blockDim.x; s *= 2)
    {
        if ((tid % (2*s)) == 0)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__
void reduce2(const int* const d_idata, int* const d_odata){
    extern __shared__ int sdata[];
    // __shared__ int sdata[TILE_WIDTH];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = d_idata[i];
    __syncthreads();
    
    for (unsigned int s=1; s < blockDim.x ; s*=2){
        int index = 2 * s* tid;
        if(index < blockDim.x){
            sdata[index] += sdata[index+s];
        }
        __syncthreads();
    }
    if(tid == 0)
        d_odata[blockIdx.x] = sdata[0];
}

__global__
void reduce3(const int* const d_idata, int* const d_odata){
    extern __shared__ int sdata[];
    // __shared__ int sdata[TILE_WIDTH];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = d_idata[i];
    __syncthreads();
    
    for (unsigned int s=blockDim.x/2; s > 0 ; s>>=1){
        if(tid < s){
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    if(tid == 0)
        d_odata[blockIdx.x] = sdata[0];
}

__global__
void reduce4(const int* const d_idata, int* const d_odata){
    extern __shared__ int sdata[];
    // __shared__ int sdata[TILE_WIDTH];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;
    sdata[tid] = d_idata[i] + d_idata[i+blockDim.x];
    __syncthreads();
    
    for (unsigned int s=blockDim.x/2; s > 0 ; s>>=1){
        if(tid < s){
            sdata[tid] += sdata[tid+s];
        }
        __syncthreads();
    }
    if(tid == 0)
        d_odata[blockIdx.x] = sdata[0];
}

__device__ void unroll(volatile int* sdata, int t) {
	sdata[t] += sdata[t + 32];
	sdata[t] += sdata[t + 16];
	sdata[t] += sdata[t + 8];
	sdata[t] += sdata[t + 4];
	sdata[t] += sdata[t + 2];
	sdata[t] += sdata[t + 1];
}


__global__ void reduce5(const int* const d_idata, int* const d_odata) {
	__shared__ int sdata[SM_SIZE];

	int tid = threadIdx.x;

	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdata[tid] = d_idata[i] + d_idata[i + blockDim.x];
	__syncthreads();
	for (int s = blockDim.x / 2; s > 32; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid < 32) {
		unroll(sdata, tid);
	}

	if (tid == 0) {
		d_odata[blockIdx.x] = sdata[0];
	}
}


template <unsigned int blockSize>
__global__ void reduce6 (const int* const d_idata, int* const d_odata)
{
	extern __shared__ int sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	sdata[threadIdx.x] = d_idata[i] + d_idata[i + blockDim.x];
	__syncthreads();
	for (int s = blockDim.x / 2; s > 32; s >>= 1) {
		if (threadIdx.x < s) {
			sdata[threadIdx.x] += sdata[threadIdx.x + s];
		}
		__syncthreads();
	}
    if (blockSize >= 512) {if (tid < 256) { sdata[tid] = (sdata[tid] + sdata[tid+256]);} __syncthreads(); } 
    if (blockSize >= 256) {if (tid < 128) { sdata[tid] = (sdata[tid] + sdata[tid+128]);} __syncthreads(); } 
    if (blockSize >= 128) {if (tid < 64) {  sdata[tid] = (sdata[tid] + sdata[tid+64]);}  __syncthreads(); } 

    if (tid < 32){
        if(blockSize >= 64) {
             sdata[tid] = (sdata[tid] + sdata[tid+32]);
        }
            __syncthreads();
        if(blockSize >= 32){
             sdata[tid] = (sdata[tid] + sdata[tid+16]);
        } 
            __syncthreads();
        if(blockSize >= 16){
             sdata[tid] = (sdata[tid] + sdata[tid+8]);
        } 
            __syncthreads();
        if (blockSize >= 8){
             sdata[tid] = (sdata[tid] + sdata[tid+4]);
        }
            __syncthreads();
        if(blockSize >= 4)  {
            sdata[tid] = (sdata[tid] + sdata[tid+2]);
        }
            __syncthreads();
        if (blockSize >= 2){
            sdata[tid] = (sdata[tid] + sdata[tid+1]);
        } 
            __syncthreads();
    }
    if (tid == 0) d_odata[blockIdx.x] = sdata[0];
}

void reduce6_switch(const int* const d_idata, int* const d_odata, int block_num, int blockdim ){
    std::cout << "cuda block_num: " << block_num << std::endl;
    switch(block_num){
        case 512:
            reduce6<512><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);          break;
        case 256:
            reduce6<256><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);          break;
        case 128:
            reduce6<128><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);          break;
        case 64:
            reduce6<64><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);           break;
        case 32:
            reduce6<32><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);           break;
        case 16:
            reduce6<16><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);           break;
        case 8:
            reduce6<8><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);            break;
        case 4:
            reduce6<4><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);            break;
        case 2:
            reduce6<2><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);            break;
        case 1:
            reduce6<1><<<block_num, blockdim, SM_SIZE>>>(d_idata, d_odata);            break;
    }
}

// void reduce_optimize(const int* const g_idata, int* const g_odata, const int* const d_idata, int* const d_odata, const int n) {
//     int size = n;
//     int block_num = 256;
//     int blockdim = ((size-1) / block_num) +1; 
//     // int blockdim = ((size-1) / block_num)/2 +1; 
//     reduce1<<< blockdim, block_num, SM_SIZE >>>(d_idata, d_odata);
//     for (int i = block_num ; i >= block_num; i /=2){
//         std::cout << "i: " << i << std::endl;
//         reduce1<<< blockdim, i, SM_SIZE >>>(d_odata, d_odata);
//     }
//     reduce1<<< 1, block_num, SM_SIZE >>>(d_odata, d_odata);
// }

void reduce_optimize(const int* const g_idata, int* const g_odata, const int* const d_idata, int* const d_odata, const int n) {
    int size = n;
    int block_dim = 256;
    // int block_num = ((size-1) / block_dim) +1; 
    int block_num = ((size-1) / block_dim)/2 +1;
    cout <<  "block_num: " << block_num << endl;
    // reduce5<<< block_num, block_dim, SM_SIZE >>>(d_idata, d_odata);
    // for (int i = block_num/2 ; i >= block_dim; i =sqrt(i)){
    //     std::cout << "i: " << i << std::endl;
    //     reduce5<<< i, block_dim, SM_SIZE >>>(d_odata, d_odata);
    // }
    // reduce5<<< 1, block_dim, SM_SIZE >>>(d_odata, d_odata);
}